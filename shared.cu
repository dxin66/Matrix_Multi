#include <hip/hip_runtime.h>

#include <iostream>

// CUDA错误检查宏
#define CUDA_CHECK_ERROR(err) \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(-1); \
    }
    
__global__ void matrixMulSharedKernel(float* fpMatrixA, float* fpMatrixB,float* fpMatrixC, int m, int n, int k)
{
    int nRow = blockIdx.y * blockDim.y + threadIdx.y;
    int nCol = blockIdx.x * blockDim.x + threadIdx.x;
    float fCVal = 0.0f;

    int BLOCK_SIZE=32;
    __shared__ float shTileA[32][32];
    __shared__ float shTileB[32][32];

    int nIter = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    for(int i = 0; i < nIter; i++)
    {
        // load data from global memory to shared memory
        shTileA[threadIdx.y][threadIdx.x] = fpMatrixA[nRow * k + i * BLOCK_SIZE + threadIdx.x];
        shTileB[threadIdx.y][threadIdx.x] = fpMatrixB[(i * BLOCK_SIZE + threadIdx.y) * n + nCol];

        // sync to wait for all threads in one block to finish loading datas
        __syncthreads();

        // sub-matrix multiply
        for(int l = 0; l < BLOCK_SIZE; l++)
        {
            fCVal += shTileA[threadIdx.y][l] * shTileB[l][threadIdx.x];
        }

        // sync to wait for all threads in one block to finish compute
        __syncthreads();
    }

    // store results into global memory
    fpMatrixC[nRow * n + nCol] = fCVal;
}
int main() {
    // 设置矩阵维度和分配内存
    int m =2000;       //矩阵A的行数
    int n = 2000;       //矩阵B的列数
    int k = 2000; // 共享维度
    int sizeA = m * k * sizeof(float);
    int sizeB = k * n * sizeof(float);
    int sizeC = m * n * sizeof(float);

    float* d_A, * d_B, * d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    // 初始化主机端矩阵
    float* hostMatrixA = new float[sizeA];
    float* hostMatrixB = new float[sizeB];
    float* hostMatrixC = new float[sizeC];

    for (int i = 0; i < sizeA; i++) {
        hostMatrixA[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    for (int i = 0; i < sizeB; i++) {
        hostMatrixB[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // 将矩阵数据从主机复制到设备
    hipMemcpy(d_A, hostMatrixA, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, hostMatrixB, sizeB, hipMemcpyHostToDevice);

    // 配置线程块和网格
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((n + threadsPerBlock.x - 1) / threadsPerBlock.x, (m + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // 创建CUDA事件
    hipEvent_t start, stop;
    CUDA_CHECK_ERROR(hipEventCreate(&start));
    CUDA_CHECK_ERROR(hipEventCreate(&stop));

    // 记录开始时间
    CUDA_CHECK_ERROR(hipEventRecord(start));

    // 执行GPU核函数
    matrixMulSharedKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, m, n, k);

    // 记录结束时间
    CUDA_CHECK_ERROR(hipEventRecord(stop));
    CUDA_CHECK_ERROR(hipEventSynchronize(stop));

    // 将结果从设备复制回主机
    hipMemcpy(hostMatrixC, d_C, sizeC, hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // 计算执行时间
    float milliseconds = 0;
    CUDA_CHECK_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "GPU Execution Time: " << milliseconds << " ms" << std::endl;

    // 释放主机内存
    delete[] hostMatrixA;
    delete[] hostMatrixB;
    delete[] hostMatrixC;

    return 0;
}